#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "task11.h"
#include <hip/device_functions.h>
#include <stdio.h>
#include <fstream>
#include <limits.h>
#include <ctime> 

int maxThreadsDim[3];

struct ExecutionInfo
{
	Task11::Result res;
	float time; //sec
};

__global__ void findSubcolumnSum(int* matrix, int* columnsSums, int rows, int columns, int subrows)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int row = i / columns;
	int column = i % columns;

	if (row >= rows || column >= columns) return;

	int sum = 0;

	for (int curRow = row; curRow < row + subrows; curRow++)
	{
		sum += matrix[columns * curRow + column];
	}

	columnsSums[columns * row + column] = sum;
}

__device__ int maxSum = INT_MIN;
__device__ int submatrixFirstElementId;

__global__ void findMaxSubcolumnSum(int* columnsSums, int arrayRows, int arrayColumns, int subColumns)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	if (row >= arrayRows) return;

	int threadMaxSum = 0, sum = 0, firstElementId = row * arrayColumns;

	for (int i = 0; i < subColumns; i++)
	{
		sum += columnsSums[arrayColumns * row + i];
	}

	threadMaxSum = sum;

	for (int i = 0; i < arrayColumns - subColumns; i++)
	{
		sum -= columnsSums[arrayColumns * row + i];
		sum += columnsSums[arrayColumns * row + i + subColumns];

		if (threadMaxSum < sum)
		{
			threadMaxSum = sum;
			firstElementId = row * arrayColumns + i + 1;
		}
	}

	atomicMax(&maxSum, threadMaxSum);
	__syncthreads();

	if (maxSum == threadMaxSum)
		atomicExch(&submatrixFirstElementId, firstElementId);
}

void getNumberOfBlocksAndThreads(int elemsCount, int* blocks, int* threads)
{
	*blocks = (elemsCount - 1) / maxThreadsDim[0] + 1;
	*threads = (elemsCount < maxThreadsDim[0] ? elemsCount : maxThreadsDim[0]);
}

hipError_t allocateMemory(int* matrix, int** gpuMatrix, int rows, int columns, int** gpuColumnsSums, int subRows)
{
	hipError_t status = hipMalloc(gpuMatrix, rows * columns * sizeof(int));
	status = hipMemcpy(*gpuMatrix, matrix, rows * columns * sizeof(int), hipMemcpyHostToDevice);
	status = hipMalloc(gpuColumnsSums, (rows - subRows + 1) * columns * sizeof(int));
	return status;
}

Task11::Result findSubmatrixWithMaxSumCuda(int* matrix, int rows, int columns, int subrows, int subcolumns)
{
	if (matrix == nullptr || subrows > rows || subcolumns > columns ||
		rows < 0 || columns < 0 || subrows < 0 || subcolumns < 0)
		throw std::invalid_argument::exception();

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	memcpy(maxThreadsDim, deviceProp.maxThreadsDim, 3 * sizeof(int));

	int* _matrix = nullptr, *subcolumnsSums = nullptr;
	dim3 subcolumnsDim = { (unsigned int)rows - subrows + 1 , (unsigned int)columns };

	if (allocateMemory(matrix, &_matrix, rows, columns, &subcolumnsSums, subrows) != hipSuccess)
	{
		printf("Allocate memory error!\n");
		return { -1,-1 };
	}

	int blocks, threads;
	getNumberOfBlocksAndThreads(subcolumnsDim.x * subcolumnsDim.y, &blocks, &threads);

	findSubcolumnSum << < blocks, threads >> > (_matrix, subcolumnsSums, rows, columns, subrows);
	hipDeviceSynchronize();

	getNumberOfBlocksAndThreads(subcolumnsDim.x, &blocks, &threads);

	findMaxSubcolumnSum << < blocks, threads >> > (subcolumnsSums, subcolumnsDim.x, subcolumnsDim.y, subcolumns);
	hipDeviceSynchronize();

	Task11::Result res;
	hipMemcpyFromSymbol(&res, HIP_SYMBOL(maxSum), sizeof(int));
	hipMemcpyFromSymbol(&res.firstElementId, HIP_SYMBOL(submatrixFirstElementId), sizeof(int));

	hipFree(_matrix);
	hipFree(subcolumnsSums);
	return res;
}

void generateMatrix(int** matrix, int rows, int columns, int min, int max)
{
	if (rows <= 0 || columns <= 0) throw std::invalid_argument::exception();

	srand(time(0));

	*matrix = (int*)malloc(rows * columns * sizeof(int));

	for (size_t i = 0; i < rows; ++i)
	{
		for (size_t j = 0; j < columns; ++j)
		{
			(*matrix)[i * columns + j] = rand () % (max - min + 1) + min;
		}
	}
}

void readInfoFromFile(int& rows, int& columns, int& min, int& max, int& subrows, int& subcolumns)
{
	std::ifstream fin("info.txt");
	if (!fin) return;

	fin >> rows >> columns >> min >> max >> subrows >> subcolumns;
	fin.close();
}

/*
void readMatrixFromFile(int** matrix, int& rows, int& columns)
{
	std::ifstream fin("matrix.txt");
	if (!fin) return;

	fin >> rows >> columns;
	*matrix = (int*)malloc(rows * columns * sizeof(int));

	for (size_t i = 0; i < rows; ++i)
	{
		for (size_t j = 0; j < columns; ++j)
		{
			fin >> (*matrix)[i * columns + j];
		}
	}

	fin.close();
}
*/

void writeResultsToFile(int* matrix, int rows, int columns, int subrows, int subcolumns, ExecutionInfo withCuda, ExecutionInfo withoutCuda)
{
	std::ofstream fout("result.txt");
	if (!fout) return;

	auto writeSubmatrix = [&fout, matrix, rows, columns, subrows, subcolumns](Task11::Result res)
	{
		int startRow = res.firstElementId / columns;
		int startColumn = res.firstElementId % columns;

		for (int i = startRow; i < startRow + subrows; i++)
		{
			for (int j = startColumn; j < startColumn + subcolumns; j++)
				fout << matrix[i * columns + j] << " ";

			fout << "\n";
		}
	};

	fout << "The results are " << (withCuda.res == withoutCuda.res ? "" : "not ") << "equal\n";

	fout << "\nWith cuda:\n" << "Time: " << withCuda.time << "c\n" 
		 << "Sum: " << withCuda.res.sum << "\n" << "Submatrix:\n";
	writeSubmatrix(withCuda.res);

	fout << "\nWithout cuda:\n" << "Time: " << withoutCuda.time << "c\n"
		 << "Sum: " << withoutCuda.res.sum << "\n" << "Submatrix:\n";
	writeSubmatrix(withoutCuda.res);

	fout.close();
}

int main(int argc, char** argv)
{
	int* matrix;

	int rows, columns, min, max, subrows, subcolumns;
	readInfoFromFile(rows, columns, min, max, subrows, subcolumns);

	//readMatrixFromFile(&matrix, rows, columns);
	generateMatrix(&matrix, rows, columns, min, max);

	ExecutionInfo info1, info2;
	struct timespec start, end;

	clock_gettime(CLOCK_REALTIME, &start);
	//unsigned int start_time = clock();
	info1.res = findSubmatrixWithMaxSumCuda(matrix, rows, columns, subrows, subcolumns);
	//unsigned int end_time = clock();
	//info1.time = (float)(end_time - start_time) / CLOCKS_PER_SEC;
	clock_gettime(CLOCK_REALTIME, &end);
	info1.time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) * 1.0 / 1000000000;


	clock_gettime(CLOCK_REALTIME, &start);
	//start_time = clock();
	info2.res = Task11::findSubmatrixWithMaxSum(matrix, rows, columns, subrows, subcolumns);
	//end_time = clock();
	//info2.time = (float)(end_time - start_time) / CLOCKS_PER_SEC;
	clock_gettime(CLOCK_REALTIME, &end);
	info2.time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) * 1.0 / 1000000000;

	writeResultsToFile(matrix, rows, columns, subrows, subcolumns, info1, info2);
	return 0;
}