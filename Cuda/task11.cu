#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdexcept>
#include "task11.h"

using namespace Task11;

int maxThreadsDim[3];

__global__ void findSubcolumnSum(int* matrix, int* columnsSums, int rows, int columns, int subrows)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int column = i % columns;

	if (column >= columns) return;

	int sum = 0;

	for (int curRow = 0; curRow < subrows; curRow++)
	{
		sum += matrix[columns * curRow + column];
	}
	columnsSums[column] = sum;

	for (int curRow = subrows; curRow < rows; curRow++)
	{
		sum -= matrix[columns * (curRow - subrows) + column];
		sum += matrix[columns * curRow + column];

		columnsSums[columns * (curRow - subrows + 1) + column] = sum;
	}
}

__device__ int maxSum = INT_MIN;
__device__ int submatrixFirstElementId;

__global__ void findMaxSumOfSubcolumns(int* columnsSums, int arrayRows, int arrayColumns, int subColumns)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	if (row >= arrayRows) return;

	int threadMaxSum, sum = 0;
	int firstElementId = row * arrayColumns;

	for (int i = 0; i < subColumns; i++)
	{
		sum += columnsSums[arrayColumns * row + i];
	}

	threadMaxSum = sum;

	for (int i = 0; i < arrayColumns - subColumns; i++)
	{
		sum -= columnsSums[arrayColumns * row + i];
		sum += columnsSums[arrayColumns * row + i + subColumns];

		if (threadMaxSum < sum)
		{
			threadMaxSum = sum;
			firstElementId = row * arrayColumns + i + 1;
		}
	}

	atomicMax(&maxSum, threadMaxSum);
	__syncthreads();

	if (maxSum == threadMaxSum)
		atomicExch(&submatrixFirstElementId, firstElementId);
}

void getNumberOfBlocksAndThreads(int elemsCount, int* blocks, int* threads)
{
	*blocks = (elemsCount - 1) / maxThreadsDim[0] + 1;
	*threads = (elemsCount < maxThreadsDim[0] ? elemsCount : maxThreadsDim[0]);
}

hipError_t allocateMemory(int* matrix, int** gpuMatrix, int rows, int columns, int** gpuColumnsSums, int subRows)
{
	hipError_t status = hipMalloc(gpuMatrix, rows * columns * sizeof(int));
	status = hipMemcpy(*gpuMatrix, matrix, rows * columns * sizeof(int), hipMemcpyHostToDevice);
	status = hipMalloc(gpuColumnsSums, (rows - subRows + 1) * columns * sizeof(int));
	return status;
}

Result Cuda::findSubmatrixWithMaxSum(int* matrix, int rows, int columns, int subrows, int subcolumns)
{
	if (matrix == nullptr || subrows > rows || subcolumns > columns ||
		rows < 0 || columns < 0 || subrows < 0 || subcolumns < 0)
		throw std::invalid_argument::exception();

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	memcpy(maxThreadsDim, deviceProp.maxThreadsDim, 3 * sizeof(int));

	int* _matrix = nullptr, * subcolumnsSums = nullptr;
	dim3 subcolumnsDim = { (unsigned int)rows - subrows + 1 , (unsigned int)columns };

	if (allocateMemory(matrix, &_matrix, rows, columns, &subcolumnsSums, subrows) != hipSuccess)
	{
		printf("Allocate memory error!\n");
		return { -1,-1 };
	}

	int blocks, threads;

	getNumberOfBlocksAndThreads(subcolumnsDim.y, &blocks, &threads);
	findSubcolumnSum << <blocks, threads >> > (_matrix, subcolumnsSums, rows, columns, subrows);

	hipDeviceSynchronize();

	getNumberOfBlocksAndThreads(subcolumnsDim.x, &blocks, &threads);
	findMaxSumOfSubcolumns << < blocks, threads >> > (subcolumnsSums, subcolumnsDim.x, subcolumnsDim.y, subcolumns);

	hipDeviceSynchronize();

	Task11::Result res;
	hipMemcpyFromSymbol(&res.sum, HIP_SYMBOL(maxSum), sizeof(int));
	hipMemcpyFromSymbol(&res.firstElementId, HIP_SYMBOL(submatrixFirstElementId), sizeof(int));

	hipFree(_matrix);
	hipFree(subcolumnsSums);
	return res;
}
