#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdexcept>
#include "task11.h"

using namespace Task11;

int maxThreadsDim[3];

__global__ void findSubcolumnSum(long long* matrix, long long* columnsSums, int rows, int columns, int subrows)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int column = i % columns;

	if (column >= columns) return;

	int sum = 0;

	for (int curRow = 0; curRow < subrows; curRow++)
	{
		sum += matrix[columns * curRow + column];
	}
	columnsSums[column] = sum;

	for (int curRow = subrows; curRow < rows; curRow++)
	{
		sum -= matrix[columns * (curRow - subrows) + column];
		sum += matrix[columns * curRow + column];

		columnsSums[columns * (curRow - subrows + 1) + column] = sum;
	}
}

__device__ long long maxSum = INT_MIN;
__device__ int submatrixFirstElementId;

__global__ void findMaxSumOfSubcolumns(long long* columnsSums, int arrayRows, int arrayColumns, int subColumns)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	if (row >= arrayRows) return;

	int threadMaxSum = 0, sum = 0, firstElementId = row * arrayColumns;

	for (int i = 0; i < subColumns; i++)
	{
		sum += columnsSums[arrayColumns * row + i];
	}

	threadMaxSum = sum;

	for (int i = 0; i < arrayColumns - subColumns; i++)
	{
		sum -= columnsSums[arrayColumns * row + i];
		sum += columnsSums[arrayColumns * row + i + subColumns];

		if (threadMaxSum < sum)
		{
			threadMaxSum = sum;
			firstElementId = row * arrayColumns + i + 1;
		}
	}

	atomicMax(&maxSum, threadMaxSum);
	__syncthreads();

	if (maxSum == threadMaxSum)
		atomicExch(&submatrixFirstElementId, firstElementId);
}

void getNumberOfBlocksAndThreads(int elemsCount, int* blocks, int* threads)
{
	*blocks = (elemsCount - 1) / maxThreadsDim[0] + 1;
	*threads = (elemsCount < maxThreadsDim[0] ? elemsCount : maxThreadsDim[0]);
}

hipError_t allocateMemory(long long* matrix, long long** gpuMatrix, int rows, int columns, long long** gpuColumnsSums, int subRows)
{
	hipError_t status = hipMalloc(gpuMatrix, rows * columns * sizeof(long long));
	status = hipMemcpy(*gpuMatrix, matrix, rows * columns * sizeof(long long), hipMemcpyHostToDevice);
	status = hipMalloc(gpuColumnsSums, (rows - subRows + 1) * columns * sizeof(long long));
	return status;
}

Result Cuda::findSubmatrixWithMaxSum(long long* matrix, int rows, int columns, int subrows, int subcolumns)
{
	if (matrix == nullptr || subrows > rows || subcolumns > columns ||
		rows < 0 || columns < 0 || subrows < 0 || subcolumns < 0)
		throw std::invalid_argument::exception();

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	memcpy(maxThreadsDim, deviceProp.maxThreadsDim, 3 * sizeof(int));

	long long* _matrix = nullptr, * subcolumnsSums = nullptr;
	dim3 subcolumnsDim = { (unsigned int)rows - subrows + 1 , (unsigned int)columns };

	if (allocateMemory(matrix, &_matrix, rows, columns, &subcolumnsSums, subrows) != hipSuccess)
	{
		printf("Allocate memory error!\n");
		return { -1,-1 };
	}

	int blocks, threads;

	getNumberOfBlocksAndThreads(subcolumnsDim.y, &blocks, &threads);
	findSubcolumnSum << <blocks, threads >> > (_matrix, subcolumnsSums, rows, columns, subrows);

	hipDeviceSynchronize();

	getNumberOfBlocksAndThreads(subcolumnsDim.x, &blocks, &threads);
	findMaxSumOfSubcolumns << < blocks, threads >> > (subcolumnsSums, subcolumnsDim.x, subcolumnsDim.y, subcolumns);

	hipDeviceSynchronize();

	Task11::Result res;
	hipMemcpyFromSymbol(&res, HIP_SYMBOL(maxSum), sizeof(long long));
	hipMemcpyFromSymbol(&res.firstElementId, HIP_SYMBOL(submatrixFirstElementId), sizeof(int));

	hipFree(_matrix);
	hipFree(subcolumnsSums);
	return res;
}