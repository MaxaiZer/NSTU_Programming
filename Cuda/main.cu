#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "task11.h"
#include <hip/device_functions.h>
#include <stdio.h>
#include <fstream>
#include <limits.h>
#include <ctime> 

struct ExecutionInfo
{
	Task11::Result res;
	float time; //sec
};

void generateMatrix(int** matrix, int rows, int columns, int min, int max)
{
	if (rows <= 0 || columns <= 0) throw std::invalid_argument::exception();

	srand(time(0));

	*matrix = (int*)malloc(rows * columns * sizeof(int));

	for (size_t i = 0; i < rows; ++i)
	{
		for (size_t j = 0; j < columns; ++j)
		{
			(*matrix)[i * columns + j] = rand() % (max - min + 1) + min;
		}
	}
}

void readInfoFromFile(int& rows, int& columns, int& min, int& max, int& subrows, int& subcolumns)
{
	std::ifstream fin("info.txt");
	if (!fin) return;

	fin >> rows >> columns >> min >> max >> subrows >> subcolumns;
	fin.close();
}

void writeResultsToFile(int* matrix, int rows, int columns, int subrows, int subcolumns, ExecutionInfo withCuda, ExecutionInfo withoutCuda)
{
	std::ofstream fout("result.txt");
	if (!fout) return;

	auto writeSubmatrix = [&fout, matrix, rows, columns, subrows, subcolumns](Task11::Result res)
	{
		int startRow = res.firstElementId / columns;
		int startColumn = res.firstElementId % columns;

		for (int i = startRow; i < startRow + subrows; i++)
		{
			for (int j = startColumn; j < startColumn + subcolumns; j++)
				fout << matrix[i * columns + j] << " ";

			fout << "\n";
		}
	};

	fout << "The results are " << (withCuda.res == withoutCuda.res ? "" : "not ") << "equal\n";

	fout << "\nWith cuda:\n" << "Time: " << withCuda.time << "c\n"
		<< "Sum: " << withCuda.res.sum << "\n" << "Submatrix:\n";
	writeSubmatrix(withCuda.res);

	fout << "\nWithout cuda:\n" << "Time: " << withoutCuda.time << "c\n"
		<< "Sum: " << withoutCuda.res.sum << "\n" << "Submatrix:\n";
	writeSubmatrix(withoutCuda.res);

	fout.close();
}

int main(int argc, char** argv)
{
	int* matrix;

	int rows, columns, min, max, subrows, subcolumns;
	readInfoFromFile(rows, columns, min, max, subrows, subcolumns);

	generateMatrix(&matrix, rows, columns, min, max);

	ExecutionInfo info1, info2;
	struct timespec start, end;

	clock_gettime(CLOCK_REALTIME, &start);
	//unsigned int start_time = clock();
	info1.res = Task11::Cuda::findSubmatrixWithMaxSum(matrix, rows, columns, subrows, subcolumns);
	//unsigned int end_time = clock();
	//info1.time = (float)(end_time - start_time) / CLOCKS_PER_SEC;
	clock_gettime(CLOCK_REALTIME, &end);
	info1.time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) * 1.0 / 1000000000;


	clock_gettime(CLOCK_REALTIME, &start);
	//start_time = clock();
	info2.res = Task11::NoCuda::findSubmatrixWithMaxSum(matrix, rows, columns, subrows, subcolumns);
	//end_time = clock();
	//info2.time = (float)(end_time - start_time) / CLOCKS_PER_SEC;
	clock_gettime(CLOCK_REALTIME, &end);
	info2.time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) * 1.0 / 1000000000;

	writeResultsToFile(matrix, rows, columns, subrows, subcolumns, info1, info2);
	return 0;
}