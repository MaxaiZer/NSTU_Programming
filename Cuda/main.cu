#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "task11.h"
#include <hip/device_functions.h>
#include <stdio.h>
#include <fstream>
#include <limits.h>
#include <ctime> 

struct ExecutionInfo
{
	Task11::Result res;
	float time; //sec
};

void generateMatrix(long long** matrix, int rows, int columns, int min, int max)
{
	if (rows <= 0 || columns <= 0) throw std::invalid_argument::exception();

	srand(time(0));

	*matrix = (long long*)malloc(rows * columns * sizeof(long long));

	for (size_t i = 0; i < rows; ++i)
	{
		for (size_t j = 0; j < columns; ++j)
		{
			(*matrix)[i * columns + j] = rand() % (max - min + 1) + min;
		}
	}
}

void readInfoFromFile(int& rows, int& columns, int& min, int& max, int& subrows, int& subcolumns)
{
	std::ifstream fin("info.txt");
	if (!fin) return;

	fin >> rows >> columns >> min >> max >> subrows >> subcolumns;
	fin.close();
}

void writeResultsToFile(long long* matrix, int rows, int columns, int subrows, int subcolumns, ExecutionInfo withCuda, ExecutionInfo withoutCuda)
{
	std::ofstream fout("result.txt");
	if (!fout) return;

	auto writeSubmatrix = [&fout, matrix, rows, columns, subrows, subcolumns](Task11::Result res)
	{
		int startRow = res.firstElementId / columns;
		int startColumn = res.firstElementId % columns;

		for (int i = startRow; i < startRow + subrows; i++)
		{
			for (int j = startColumn; j < startColumn + subcolumns; j++)
				fout << matrix[i * columns + j] << " ";

			fout << "\n";
		}
	};

	fout << "The results are " << (withCuda.res == withoutCuda.res ? "" : "not ") << "equal\n";

	fout << "\nWith cuda:\n" << "Time: " << withCuda.time << "c\n"
		<< "Sum: " << withCuda.res.sum << "\n" << "Submatrix:\n";
	writeSubmatrix(withCuda.res);

	fout << "\nWithout cuda:\n" << "Time: " << withoutCuda.time << "c\n"
		<< "Sum: " << withoutCuda.res.sum << "\n" << "Submatrix:\n";
	writeSubmatrix(withoutCuda.res);

	fout.close();
}

int main(int argc, char** argv)
{
	long long* matrix;

	int rows, columns, min, max, subrows, subcolumns;
	readInfoFromFile(rows, columns, min, max, subrows, subcolumns);

	generateMatrix(&matrix, rows, columns, min, max);

	auto execute = [matrix, rows, columns, subrows, subcolumns](Task11::Result(*func)(long long*, int,int,int,int)) 
	{        
		ExecutionInfo info;
		struct timespec start, end;
		clock_gettime(CLOCK_REALTIME, &start);
		info.res = func(matrix, rows, columns, subrows, subcolumns);
		clock_gettime(CLOCK_REALTIME, &end);
		info.time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) * 1.0 / 1000000000;
		return info;
	};

	ExecutionInfo info1 = execute(Task11::Cuda::findSubmatrixWithMaxSum);
	ExecutionInfo info2 = execute(Task11::NoCuda::findSubmatrixWithMaxSum);

	writeResultsToFile(matrix, rows, columns, subrows, subcolumns, info1, info2);
	return 0;
}